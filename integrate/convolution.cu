#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"
#define BLOCK_SIZE 16
#define CWIDTH 16
#define CHEIGHT 16
#define FWIDTH 3
#define FHEIGHT 3
#define CNUMBER 16
#define COUT 16

__shared__ float filteroutshared[CHEIGHT][CNUMBER][CNUMBER];

__global__ void convolute_kernel(float *IC, float*F, float*FP, float *SOC) {



    int idx = threadIdx.x;
    int idy = threadIdx.y;

    //printf("hello\n");
    int kCenterX = FWIDTH / 2;
    int kCenterY = FHEIGHT / 2;
    int filtersperthread = CNUMBER / BLOCK_SIZE;
    int remaining = CNUMBER % BLOCK_SIZE;
    float filtertemp[FHEIGHT * FWIDTH];
    float outputtemp[(CHEIGHT / BLOCK_SIZE) * CWIDTH];
    //printf("fil per t %d",filtersperthread);
    //register float filtertemp[FHEIGHT * FWIDTH];
    //register float outputtemp[CHEIGHT / BLOCK_SIZE * CWIDTH];
    //printf("hello");
    int chunk_size = CHEIGHT / BLOCK_SIZE;
    int ilimits = idy * chunk_size;
    int ioffset;
    int joffset;
    int myoffset;
    int kchanneloffset;
    int moffset;
    int i_tmp;
    int mm,nn;
    int test;
    int iioffset;
    float sepFilter[CNUMBER];
  
    for(int i = 0 ; i < CNUMBER; i++)
        sepFilter[i] = FP[idx * CNUMBER + i];
    for(int k = 0; k < filtersperthread; k++)
    {
        kchanneloffset = k * BLOCK_SIZE;
        for(int i = 0; i < FHEIGHT; i++)
        {
            moffset = i * FWIDTH;
            ioffset = i * CNUMBER * FWIDTH;
            myoffset = kchanneloffset + ioffset + idx;            
            //ioffset = (k * FHEIGHT * FWIDTH * BLOCK_SIZE) + (idx * FHEIGHT * FWIDTH)
            for(int j = 0; j < FWIDTH; j++)
            {
                joffset = j * CNUMBER;
                filtertemp[moffset + j] = F[myoffset + joffset];
                //printf("%.1f    %.1f\n",filtertemp[k][i][j],F[k * FHEIGHT * FWIDTH * BLOCK_SIZE + idx%BLOCK_SIZE * FHEIGHT * FWIDTH + i * FWIDTH + j]); 

            }
        }


        for(int i = ilimits - kCenterY; i < ilimits + chunk_size + kCenterY; i++)
        {
            if(i >= 0 && i < CHEIGHT)
            {
                ioffset = i * CNUMBER * CWIDTH;
                myoffset = kchanneloffset + ioffset + idx;
                for(int j = 0; j < CWIDTH; j++)
                {
                    joffset = j * CNUMBER;
                    int input_element = IC[myoffset + joffset];

                    for (int m = 0; m < FHEIGHT; ++m)     // kernel rows
                    {
                        mm = FHEIGHT - 1 - m;
                        int ii = (i - kCenterY) + mm;
                        iioffset = ((ii - ilimits) * CWIDTH);
                        moffset = m * FWIDTH;

                        for (int n = 0; n < FWIDTH; ++n) // kernel columns
                        {

                            nn = FWIDTH - 1 - n;
                            int jj = (j - kCenterX) + nn;
                            if ((jj >= 0 && jj < CWIDTH) && (ii >= ilimits && ii < (ilimits + chunk_size)))
                            {
                                //printf("%.1f\n", filtertemp[k][m][n] * IC[idx * CWIDTH * CWIDTH + i * CWIDTH + j]);
                                 outputtemp[iioffset + jj] += filtertemp[moffset + n] * input_element;
                            }
                                //__syncthreads();
                        }

                    }
                }
            }
        }
        /*
        for(int i = ilimits; i < ilimits + chunk_size; i++)
        {
            ioffset = i * CNUMBER * CWIDTH;
            iioffset = ((i - ilimits) * CWIDTH);
            myoffset = kchanneloffset + ioffset + idx;
            for(int j = 0; j < CWIDTH; j++)
            {
                joffset = j * CNUMBER;
                OC[myoffset + joffset] = outputtemp[iioffset + j];
                outputtemp[iioffset + j] = 0;
            }
        }
        */
    }
    
    float pointwisetemp;
    //for(int k = 0; k < BLOCK_SIZE; k++)
    //{
        //if(idy == k)
        //{
            for(int j = 0; j < CHEIGHT / BLOCK_SIZE; j++)
                for(int i = 0; i < CWIDTH; i++)
                    filteroutshared[idy][i][idx] = outputtemp[j * CWIDTH + i];
        //}
        __syncthreads();
        /*    
        if(idx == 0 && idy == 0)
        {
            for(int i = 0; i < 16; i++)
            {
                for(int j = 0; j < 16; j++)
                {
                    printf("%.1f ",filteroutshared[i][j]);
                }
                printf("\n");
            }
        }
        */
        //if(idy == k)
        //{        
            #pragma unroll
            for(int j = 0; j < COUT; j++)
            {
                pointwisetemp = 0.0;
                for(int i = 0; i < CWIDTH; i++)
                {
                    pointwisetemp += filteroutshared[idy][j][(idx + i) % CNUMBER] * sepFilter[(idx + i) % CNUMBER];
                }
                SOC[idy * COUT * CWIDTH + j * COUT + idx] = pointwisetemp;
                //if(idy == 0 && idx == 1)
                //printf("%.f  ",pointwisetemp);
            }
        //}
        __syncthreads();
    //}
    


}

/**
 * Run a simple test of matrix multiplication using CUDA
 */

void cudaconvolute(float* IC, float* F, float* FP, float* SOC, float*** OC_cpu, float*** SOC_cpu)
{
    float totalBytes_channel = sizeof(float) *  CNUMBER * CHEIGHT * CWIDTH; 
    float totalBytes_filter =  sizeof(float) *  CNUMBER * FHEIGHT * FWIDTH;
    float totalBytes_output = sizeof(float) *  COUT * CHEIGHT * CWIDTH; 
    float totalBytes_sepfilter =  sizeof(float) *  CNUMBER * COUT; 

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
   // dim3 blocks(N/BLOCK_SIZE,N/BLOCK_SIZE);

    float* device_IC;
    float* device_F;
    float* device_SOC;
    float* device_FP;

    hipMalloc(&device_IC, totalBytes_channel);
    hipMalloc(&device_F, totalBytes_filter);
    hipMalloc(&device_SOC, totalBytes_output);
    hipMalloc(&device_FP, totalBytes_sepfilter);

    hipMemcpy(device_IC, IC, totalBytes_channel, hipMemcpyHostToDevice);
    hipMemcpy(device_F, F, totalBytes_filter, hipMemcpyHostToDevice);
    hipMemcpy(device_FP, FP, totalBytes_sepfilter, hipMemcpyHostToDevice);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    double startKernelTime = CycleTimer::currentSeconds();


    hipEventRecord(start);
    convolute_kernel<<<1, threadsPerBlock>>>(device_IC, device_F, device_FP, device_SOC);
    hipEventRecord(stop);

    hipDeviceSynchronize();
    double endKernelTime = CycleTimer::currentSeconds();


    hipMemcpy(SOC, device_SOC, totalBytes_output, hipMemcpyDeviceToHost);
    

    double kernelDuration = endKernelTime - startKernelTime;
    printf("KernelDuration: %.3f ms\n", 1000.f * kernelDuration);

    float m = 0;
    hipEventElapsedTime(&m, start, stop);
    printf("CUDA Elapsed Time %f ms\n", m);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    



    bool equal = true;

    double fops = 0.0;
    for (int i = 0;i < CHEIGHT; i++){
        for (int j = 0; j < CWIDTH; j++) {
            for(int k = 0; k < COUT; k++) {
                fops += SOC[i * CWIDTH * COUT + j * COUT + k];
                if(SOC_cpu[k][i][j] != SOC[i * CWIDTH * COUT + j * COUT + k])
                {
                    equal = false;

                    //printf("%d %d %d %.1f != %.1f\n", k , i, j, SOC_cpu[k][i][j], SOC[i * CWIDTH * COUT + j * COUT + k]);
                        //break;
                }
                printf("%0.1f ",SOC[i * CWIDTH * COUT + j * COUT + k]);      
            }
            printf("\n");
        }
        printf("\n");
    }

    printf("Bandwidth = %f GFLOPS/s\n", ((2 * FHEIGHT * FWIDTH * CWIDTH * CHEIGHT * CNUMBER) + (2 * COUT * CHEIGHT * CWIDTH * CNUMBER)) / (m * 1000000));

    if(equal)
	printf("EQUAL\n");
    else
    printf("NOT EQUAL\n");
    hipFree(device_IC);
    hipFree(device_F);
    hipFree(device_SOC);
    hipFree(device_FP);
}

void
printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}

