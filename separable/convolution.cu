#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"
#define BLOCK_SIZE 16
#define CWIDTH 16
#define CHEIGHT 16
#define FWIDTH 3
#define FHEIGHT 3
#define CNUMBER 16
#define COUT 16

__shared__ float filteroutshared[CNUMBER][CNUMBER];

__global__ void convolute_kernel(float *IC, float*F, float*FP, float *SOC) {


    int idx = threadIdx.x;
    int idy = threadIdx.y;

    //printf("hello\n");
    int kCenterX = FWIDTH / 2;
    int kCenterY = FHEIGHT / 2;
    int filtersperthread = CNUMBER / BLOCK_SIZE;
    int remaining = CNUMBER % BLOCK_SIZE;

    // TODO : change these when generalizing
    float filtertemp[1][FHEIGHT][FWIDTH];
    float outputtemp[CHEIGHT / BLOCK_SIZE][CWIDTH];
    //printf("hello");

    float sepFilter[CNUMBER];
  
    for(int i = 0 ; i < CNUMBER; i++)
        sepFilter[i] = FP[idx * CNUMBER + i];

    for(int k = 0; k < filtersperthread; k++)
    {

        for(int i = 0; i < FHEIGHT; i++)
        {
            for(int j = 0; j < FWIDTH; j++)
            {

                filtertemp[k][i][j] = F[k * FHEIGHT * FWIDTH * BLOCK_SIZE + idx%BLOCK_SIZE * FHEIGHT * FWIDTH + i * FWIDTH + j];
                //printf("%.1f    %.1f\n",filtertemp[k][i][j],F[k * FHEIGHT * FWIDTH * BLOCK_SIZE + idx%BLOCK_SIZE * FHEIGHT * FWIDTH + i * FWIDTH + j]); 

            }
        }
    } 
   
    if(remaining > 0 && idx < remaining)
    {

        int k =  filtersperthread;  
        for(int i = 0; i < FHEIGHT; i++)
        {
            for(int j = 0; j < FWIDTH; j++)
            {
                filtertemp[k][i][j] = F[k * FHEIGHT * FWIDTH * BLOCK_SIZE + idx%BLOCK_SIZE * FHEIGHT * FWIDTH + i * FWIDTH + j];
            }     
        }  
    }

    /*   
    if(idx == 0 && idy == 0)
    {
        for(int i = 0; i < FHEIGHT; i++)
        {
            for(int j = 0; j < FWIDTH; j++)
            {
                printf("%.1f",filtertemp[0][i][j]);
            }
            printf("\n");
        }
    }
    */

    //idx = column number, idy = row number
    
    for(int k = 0; k < filtersperthread; k++)
    {

        for(int i = (idy * CHEIGHT / BLOCK_SIZE) - kCenterY; i < ((idy + 1) * CHEIGHT / BLOCK_SIZE) + kCenterY; i++)
        {
            for(int j = 0; j < CWIDTH; j++)
            {
                if(i >= 0 && i < CHEIGHT)
                {
                    int input_element = IC[k * BLOCK_SIZE + i * CNUMBER * CWIDTH + j * CNUMBER + idx];
                    #pragma unroll
                    for (int m = 0; m < FHEIGHT; ++m)     // kernel rows
                    {

                        for (int n = 0; n < FWIDTH; ++n) // kernel columns
                        {
                            int ii = i + (m - kCenterY);
                            int jj = j + (n - kCenterX);
                            if (ii >= 0 && ii < CHEIGHT && jj >= 0 && jj < CWIDTH && ii >= (idy * CHEIGHT / BLOCK_SIZE) && ii < ((idy + 1) * CHEIGHT / BLOCK_SIZE))
                            {
                                //printf("%.1f\n", filtertemp[k][m][n] * IC[idx * CWIDTH * CWIDTH + i * CWIDTH + j]);
                                 outputtemp[ii - (idy * CHEIGHT / BLOCK_SIZE)][jj] += filtertemp[k][m][n] * input_element;
                            }
                                //__syncthreads();
                                //if (ii >= 0 && ii < CHEIGHT && jj >= 0 && jj < CWIDTH && ii >= (idy * CHEIGHT / BLOCK_SIZE) && ii < ((idy + 1) * CHEIGHT / BLOCK_SIZE))
                                //printf("%.f\n",outputtemp[ii - (idy * CHEIGHT / BLOCK_SIZE)][jj]);
                        }
                    }
                }
            }
        }
        /*
        for(int i = (idy * CHEIGHT / BLOCK_SIZE); i < ((idy + 1) * CHEIGHT / BLOCK_SIZE); i++)
        {
            for(int j = 0; j < CWIDTH; j++)
            {
                OC[k * BLOCK_SIZE + i * CNUMBER * CWIDTH + j * CNUMBER + idx] = outputtemp[i - (idy * CHEIGHT / BLOCK_SIZE)][j];
                outputtemp[i - (idy * CHEIGHT / BLOCK_SIZE)][j] = 0;
            }
        }
        */
    }
    
    if(remaining > 0 && idx < remaining)
    {

        int k =  filtersperthread;  
        for(int i = (idy * CHEIGHT / BLOCK_SIZE) - kCenterY; i < ((idy + 1) * CHEIGHT / BLOCK_SIZE) + kCenterY; i++)
        {
            for(int j = 0; j < CWIDTH; j++)
            {
                if(i >= 0 && i < CHEIGHT)
                {
                    int input_element = IC[k * BLOCK_SIZE + i * CNUMBER * CWIDTH + j * CNUMBER + idx];
                    #pragma unroll
                    for (int m = 0; m < FHEIGHT; ++m)     // kernel rows
                    {

                        for (int n = 0; n < FWIDTH; ++n) // kernel columns
                        {
                            int ii = i + (m - kCenterY);
                            int jj = j + (n - kCenterX);
                            //if(idx == 0 && idy == 0)
                            //printf(" %d %d %d %d ", m, n, ii, jj);
                            if (ii >= 0 && ii < CHEIGHT && jj >= 0 && jj < CWIDTH && ii >= (idy * CHEIGHT / BLOCK_SIZE) && ii < ((idy + 1) * CHEIGHT / BLOCK_SIZE))
                            {
                                //printf("%f", filtertemp[k][m][n] * IC[idx * CWIDTH * CWIDTH + i * CWIDTH + j]);
                                 outputtemp[ii - (idy * CHEIGHT / BLOCK_SIZE)][jj] += filtertemp[k][m][n] * input_element;
                                // if(idx == 0 && idy == 0)
                                 //printf(" yo %d %d \n", m, n);
                            }
                                //__syncthreads();
                        }
                    }
                }
            }
        }
        /*
        for(int i = (idy * CHEIGHT / BLOCK_SIZE); i < ((idy + 1) * CHEIGHT / BLOCK_SIZE); i++)
        {
            for(int j = 0; j < CWIDTH; j++)
            {
                OC[k * BLOCK_SIZE + i * CNUMBER * CWIDTH + j * CNUMBER + idx] = outputtemp[i - (idy * CHEIGHT / BLOCK_SIZE)][j];
                outputtemp[i - (idy * CHEIGHT / BLOCK_SIZE)][j] = 0;
            }
        }
        */
    }
    
    /*    
    for(int i = 0; i < 32; i++)
        filteroutshared[idy][idx] = 0;


    for(int i = 0; i < 32; i++)
        filteroutshared[idy][(idx + i) % 32] += outputtemp[0][(idx + i) % 32];

    __syncthreads();
    if(idx == 0 && idy == 0)
    {
        for(int i = 0; i < 32; i++)
        {
            for(int j = 0; j < 32; j++)
                printf("%.1f ", filteroutshared[i][j]);
            printf("\n");
        }
        
            
    }
    */
    /*
    if(idx == 0 && idy == 5)
    {
        for(int i = (idy * CHEIGHT / BLOCK_SIZE); i < ((idy + 1) * CHEIGHT / BLOCK_SIZE); i++)
        {
            for(int j = 0; j < CWIDTH; j++)
            {
                //if (ii >= 0 && ii < CHEIGHT && jj >= 0 && jj < CWIDTH && ii >= (idy * CHEIGHT / BLOCK_SIZE) && ii < ((idy + 1) * CHEIGHT / BLOCK_SIZE))
                printf("%.f ",outputtemp[i - (idy * CHEIGHT / BLOCK_SIZE)][j]);
            }
            printf("\n");
        }
    }
    */
    
    float pointwisetemp;
    for(int k = 0; k < BLOCK_SIZE; k++)
    {
        if(idy == k)
        {
            for(int j = 0; j < CHEIGHT / BLOCK_SIZE; j++)
                for(int i = 0; i < CWIDTH; i++)
                    filteroutshared[i][idx] = outputtemp[j][i];
        }
        __syncthreads();
        /*    
        if(idx == 0 && idy == 0)
        {
            for(int i = 0; i < 16; i++)
            {
                for(int j = 0; j < 16; j++)
                {
                    printf("%.1f ",filteroutshared[i][j]);
                }
                printf("\n");
            }
        }
        */
        if(idy == k)
        {        
            #pragma unroll
            for(int j = 0; j < COUT; j++)
            {
                pointwisetemp = 0.0;
                for(int i = 0; i < CWIDTH; i++)
                {
                    pointwisetemp += filteroutshared[j][(idx + i) % CNUMBER] * sepFilter[(idx + i) % CNUMBER];
                }
                SOC[idy * COUT * CWIDTH + j * COUT + idx] = pointwisetemp;
                //if(idy == 0 && idx == 1)
                //printf("%.f  ",pointwisetemp);
            }
        }
        __syncthreads();
    }
    


}

/**
 * Run a simple test of matrix multiplication using CUDA
 */

void cudaconvolute(float* IC, float* F, float* FP, float* SOC, float*** OC_cpu, float*** SOC_cpu)
{
    float totalBytes_channel = sizeof(float) *  CNUMBER * CHEIGHT * CWIDTH; 
    float totalBytes_filter =  sizeof(float) *  CNUMBER * FHEIGHT * FWIDTH;
    float totalBytes_output = sizeof(float) *  COUT * CHEIGHT * CWIDTH; 
    float totalBytes_sepfilter =  sizeof(float) *  CNUMBER * COUT; 

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
   // dim3 blocks(N/BLOCK_SIZE,N/BLOCK_SIZE);

    float* device_IC;
    float* device_F;
    float* device_SOC;
    float* device_FP;

    hipMalloc(&device_IC, totalBytes_channel);
    hipMalloc(&device_F, totalBytes_filter);
    hipMalloc(&device_SOC, totalBytes_output);
    hipMalloc(&device_FP, totalBytes_sepfilter);

    hipMemcpy(device_IC, IC, totalBytes_channel, hipMemcpyHostToDevice);
    hipMemcpy(device_F, F, totalBytes_filter, hipMemcpyHostToDevice);
    hipMemcpy(device_FP, FP, totalBytes_sepfilter, hipMemcpyHostToDevice);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    double startKernelTime = CycleTimer::currentSeconds();


    hipEventRecord(start);
    convolute_kernel<<<1, threadsPerBlock>>>(device_IC, device_F, device_FP, device_SOC);
    hipEventRecord(stop);

    hipDeviceSynchronize();
    double endKernelTime = CycleTimer::currentSeconds();


    hipMemcpy(SOC, device_SOC, totalBytes_output, hipMemcpyDeviceToHost);
    

    double kernelDuration = endKernelTime - startKernelTime;
    printf("KernelDuration: %.3f ms\n", 1000.f * kernelDuration);

    float m = 0;
    hipEventElapsedTime(&m, start, stop);
    printf("CUDA Elapsed Time %f ms\n", m);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    



    bool equal = true;

    double fops = 0.0;
    for (int i = 0;i < CHEIGHT; i++){
        for (int j = 0; j < CWIDTH; j++) {
            for(int k = 0; k < COUT; k++) {
                fops += SOC[i * CWIDTH * COUT + j * COUT + k];
                if(SOC_cpu[k][i][j] != SOC[i * CWIDTH * COUT + j * COUT + k])
                {
                    equal = false;

                    //printf("%d %d %d %.1f != %.1f\n", k , i, j, SOC_cpu[k][i][j], SOC[i * CWIDTH * COUT + j * COUT + k]);
                        //break;
                }
                //printf("%d",OC[k * COUT * CWIDTH + i * CWIDTH + j]);      
            }
            //printf("\n");
        }
        //printf("\n");
    }

    printf("Bandwidth = %f GFLOPS/s\n", ((2 * FHEIGHT * FWIDTH * CWIDTH * CHEIGHT * CNUMBER) + (2 * COUT * CHEIGHT * CWIDTH * CNUMBER)) / (m * 1000000));

    if(equal)
	printf("EQUAL\n");
    else
    printf("NOT EQUAL\n");
    hipFree(device_IC);
    hipFree(device_F);
    hipFree(device_SOC);
    hipFree(device_FP);
}

void
printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Shared memory per block:   %d bytes\n", deviceProps.sharedMemPerBlock);
    }
    printf("---------------------------------------------------------\n");
}

